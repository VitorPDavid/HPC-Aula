
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <string.h>
#define EPSILON 1E-9
#define BLOCK_SIZE 1024
#define ALING 64


__device__ double distance( double* dx, double* dy, double* dz,
                             const double Ax,  const  double Ay, const  double Az,
                             const double Bx,  const  double By,  const  double Bz){
  
      double x = Ax - Bx;
      double y = Ay - By;
      double z = Az - Bz;

      *dx = x; *dy = y; *dz = z;
      x *= x; y *= y; z *= z;
      return 1.0 / sqrt((double)x + y + z + EPSILON);

}



__global__ void particleParticleForces_k (  double *px, double *py, double *pz,
                                            double *fx, double *fy, double *fz, double dt){
  extern __shared__  double buff[];
  double *sub_px = &buff[0],
          *sub_py = &buff[gridDim.x],
          *sub_pz = &buff[gridDim.x * 2];

  int i  = blockDim.x * blockIdx.x + threadIdx.x;
  double pX = px[i];
  double pY = py[i];
  double pZ = pz[i];

  double fX = fx[i];
  double fY = fy[i];
  double fZ = fz[i];
  
  for (int blk = 0; blk < gridDim.x; blk++){
      sub_px[threadIdx.x] = px[ blockDim.x * blk + threadIdx.x];
      sub_py[threadIdx.x] = py[ blockDim.x * blk + threadIdx.x];
      sub_pz[threadIdx.x] = pz[ blockDim.x * blk + threadIdx.x];
      __syncthreads();

    for (int j = 0; j < blockDim.x; j++){
      double dx = 0.0f,  dy = 0.0f, dz = 0.0f;
      double d  = distance(&dx, &dy, &dz, pX, pY, pZ, sub_px[j], sub_py[j], sub_pz[j]);

      fX += dx * d;
      fY += dy * d;
      fZ += dz * d;
    }
    __syncthreads();
  }

  fx[i] = fX;
  fy[i] = fY;
  fz[i] = fZ;
  
  
}

__global__ void particleParticleVelocityPosition_k (  double *px, double *py, double *pz,
                                                      double *vx, double *vy, double *vz,
                                                      double *fx, double *fy, double *fz, double dt){
  int i  = blockDim.x * blockIdx.x + threadIdx.x;
  vx[i] += dt * fx[i];
  vy[i] += dt * fy[i];
  vz[i] += dt * fz[i];

  px[i] += dt * vx[i];
  py[i] += dt * vy[i];
  pz[i] += dt * vz[i];

  
}

void particleParticle (double *px, double *py, double *pz,
                       double *vx, double *vy, double *vz,
                       double *fx, double *fy, double *fz,
                       int nParticles, int timesteps, double dt){
    int threads = BLOCK_SIZE,
    blocks  = nParticles / BLOCK_SIZE;
    if (nParticles < 1024){
      blocks = 1;
      threads = nParticles;
    }

    fprintf(stdout, "\n B(%d) T(%d) shared memory %d \n", blocks, threads, 3 * threads * sizeof(double));

    for (int t = 0; t < timesteps; t++){
      particleParticleForces_k<<<blocks, threads, 3 * threads * sizeof(double) >>>(px, py, pz, fx, fy, fz, dt);
      particleParticleVelocityPosition_k<<<blocks, threads  >>>(px, py, pz, vx, vy, vz, fx, fy, fz, dt);
      assert( hipDeviceSynchronize() == hipSuccess);
    }//end-for (int t = 0; t < timesteps; t++){

}//end-void particleParticle


//-----------------------------------------------------------------------------------------------------
void printLog(double *px, double *py, double *pz,
  double *vx, double *vy, double *vz,
  double *fx, double *fy, double *fz, int nParticles, int timestep);
void initialCondition(double *px, double *py, double *pz,
          double *vx, double *vy, double *vz,
          double *fx, double *fy, double *fz, int nParticles);
//-----------------------------------------------------------------------------------------------------
int main (int ac, char **av){
  int timesteps  = atoi(av[1]),
      nParticles = atoi(av[2]),
      flagSave = atoi(av[3]);

  double  dt =  0.00001f,
          *h_px = NULL, *h_py = NULL, *h_pz = NULL,
          *h_vx = NULL, *h_vy = NULL, *h_vz = NULL,
          *h_fx = NULL, *h_fy = NULL, *h_fz = NULL,

          *d_px = NULL, *d_py = NULL, *d_pz = NULL,
          *d_vx = NULL, *d_vy = NULL, *d_vz = NULL,
          *d_fx = NULL, *d_fy = NULL, *d_fz = NULL;


    fprintf(stdout, "\nParcile system particle to particle \n");
    fprintf(stdout, "Memory used %lu bytes \n", nParticles * sizeof(double) * 9);

    h_px = (double *) aligned_alloc(ALING, nParticles * sizeof(double));
    assert(h_px != NULL);
    h_py = (double *) aligned_alloc(ALING, nParticles * sizeof(double));
    assert(h_py != NULL);
    h_pz = (double *) aligned_alloc(ALING, nParticles * sizeof(double));
    assert(h_pz != NULL);
//-------------------------
    h_vx = (double *) aligned_alloc(ALING, nParticles * sizeof(double));
    assert(h_vx != NULL);
    h_vy = (double *) aligned_alloc(ALING, nParticles * sizeof(double));
    assert(h_vy != NULL);
    h_vz = (double *) aligned_alloc(ALING, nParticles * sizeof(double));
    assert(h_vz != NULL);

//-------------------------
    h_fx = (double *) aligned_alloc(ALING, nParticles * sizeof(double));
    assert(h_fx != NULL);
    h_fy = (double *) aligned_alloc(ALING, nParticles * sizeof(double));
    assert(h_fy != NULL);
    h_fz = (double *) aligned_alloc(ALING, nParticles * sizeof(double));
    assert(h_fz != NULL);
//-------------------------

    initialCondition(h_px, h_py, h_pz,
                     h_vx, h_vy, h_vz,
                     h_fx, h_fy, h_fz, nParticles);

    assert(hipMalloc((void**) &d_px, nParticles * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d_px, h_px, nParticles * sizeof(double),  hipMemcpyHostToDevice) == hipSuccess);

    assert(hipMalloc((void**) &d_py, nParticles * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d_py, h_py, nParticles * sizeof(double),  hipMemcpyHostToDevice) == hipSuccess);

    assert(hipMalloc((void**) &d_pz, nParticles * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d_pz, h_pz, nParticles * sizeof(double),  hipMemcpyHostToDevice) == hipSuccess);
    //-----
    assert(hipMalloc((void**) &d_vx, nParticles * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d_vx, h_vx, nParticles * sizeof(double),  hipMemcpyHostToDevice) == hipSuccess);

    assert(hipMalloc((void**) &d_vy, nParticles * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d_vy, h_vy, nParticles * sizeof(double),  hipMemcpyHostToDevice) == hipSuccess);

    assert(hipMalloc((void**) &d_vz, nParticles * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d_vz, h_vz, nParticles * sizeof(double),  hipMemcpyHostToDevice) == hipSuccess);
    //-----
    assert(hipMalloc((void**) &d_fx, nParticles * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d_fx, h_fx, nParticles * sizeof(double),  hipMemcpyHostToDevice) == hipSuccess);

    assert(hipMalloc((void**) &d_fy, nParticles * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d_fy, h_fy, nParticles * sizeof(double),  hipMemcpyHostToDevice) == hipSuccess);

    assert(hipMalloc((void**) &d_fz, nParticles * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d_fz, h_fz, nParticles * sizeof(double),  hipMemcpyHostToDevice) == hipSuccess);



    
    particleParticle(d_px, d_py, d_pz, d_vx, d_vy, d_vz, d_fx, d_fy, d_fz, nParticles, timesteps, dt);
    

    assert(hipMemcpy(h_px, d_px, nParticles * sizeof(double),  hipMemcpyDeviceToHost) == hipSuccess); 
    assert(hipMemcpy(h_py, d_py, nParticles * sizeof(double),  hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipMemcpy(h_pz, d_pz, nParticles * sizeof(double),  hipMemcpyDeviceToHost) == hipSuccess);

    assert(hipMemcpy(h_vx, d_vx, nParticles * sizeof(double),  hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipMemcpy(h_vy, d_vy, nParticles * sizeof(double),  hipMemcpyDeviceToHost) == hipSuccess); 
    assert(hipMemcpy(h_vz, d_vz, nParticles * sizeof(double),  hipMemcpyDeviceToHost) == hipSuccess);

    assert(hipMemcpy(h_fx, d_fx, nParticles * sizeof(double),  hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipMemcpy(h_fy, d_fy, nParticles * sizeof(double),  hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipMemcpy(h_fz, d_fz, nParticles * sizeof(double),  hipMemcpyDeviceToHost) == hipSuccess);

//    printLog(h_px, h_py, h_pz, h_vx, h_vy, h_vz, h_fx, h_fy, h_fz,  nParticles, timesteps);

    if (flagSave == 1)
      printLog(h_px, h_py, h_pz, h_vx, h_vy, h_vz, h_fx, h_fy, h_fz, nParticles, timesteps);

    free(h_px);free(h_py); free(h_pz);
    free(h_vx);free(h_vy); free(h_vz);
    free(h_fx);free(h_fy); free(h_fz);


    hipFree(d_px);hipFree(d_py); hipFree(d_pz);
    hipFree(d_vx);hipFree(d_vy); hipFree(d_vz);
    hipFree(d_fx);hipFree(d_fy); hipFree(d_fz);



}


/*Declarando as structs de particula e forca*/
void printLog(double *px, double *py, double *pz,
  double *vx, double *vy, double *vz,
  double *fx, double *fy, double *fz, int nParticles, int timestep){
  char fileName[128];
  sprintf(fileName, "%s-%d-log.bin", __FILE__,  timestep);
  fprintf(stdout, "Saving file [%s] ", fileName); fflush(stdout);
  FILE *ptr = fopen(fileName, "w+");
  for(int i = 0; i < nParticles; i++){
    fprintf(ptr, "%d \t %.10f %.10f %.10f \t %.10f %.10f %.10f \t %.10f %.10f %.10f \n", i,  px[i], py[i], pz[i], vx[i], vy[i], vz[i], fx[i], fy[i], fz[i]);

  }
  fclose(ptr);
  fprintf(stdout, "[OK]\n"); fflush(stdout);
}
void initialCondition(double *px, double *py, double *pz,
          double *vx, double *vy, double *vz,
          double *fx, double *fy, double *fz, int nParticles){

    srand(42);
    memset(vx, 0x00, nParticles * sizeof(double));
    memset(vy, 0x00, nParticles * sizeof(double));
    memset(vz, 0x00, nParticles * sizeof(double));

    memset(fx, 0x00, nParticles * sizeof(double));
    memset(fy, 0x00, nParticles * sizeof(double));
    memset(fz, 0x00, nParticles * sizeof(double));

    for (int i = 0; i < nParticles ; i++){
        px[i] =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
        py[i] =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
        pz[i] =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
    }//end-for (int i = 0; i < nParticles ; i++){

}