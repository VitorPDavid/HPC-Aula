
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <string.h>
#define EPSILON 1E-9
#define BLOCK_SIZE_F 512 //Work with blocks of 512 threads due to double precision - shared memory
#define BLOCK_SIZE_VP 1024 //Work with blocks of 512 threads due to double precision - shared memory
/*Declarando as structs de particula e forca*/
struct stCoord{
    double x,
          y,
          z;
};
typedef struct stCoord tpCoord;
struct stParticle
{
    tpCoord p,
            v,
            f;

};
typedef struct stParticle tpParticle;
//--------------------------------------------------------------------------------------------------------


__device__ double distance(  double* dx,
                            double* dy,
                            double* dz,
                            const tpParticle A,
                            const tpParticle B){
            double x = A.p.x - B.p.x;
            double y = A.p.y - B.p.y;
            double z = A.p.z - B.p.z;

            *dx = x; *dy = y; *dz = z;
            x *= x; y *= y; z *= z;
            return 1.0 / sqrt((double)x + y + z + EPSILON);

}


__global__ void particleParticleForces_k(tpParticle *particles, const double dt){
    extern __shared__ tpParticle subParticles[];
    int i  = blockDim.x * blockIdx.x + threadIdx.x;
//  __shared__ tpParticle subParticles[BLOCK_SIZE];
  
    for (int blk = 0; blk < gridDim.x; blk++){
        subParticles[threadIdx.x] = particles[ blockDim.x * blk + threadIdx.x];
        __syncthreads();

        for (int j = 0; j < blockDim.x; j++){
            double dx = 0.0f,  dy = 0.0f, dz = 0.0f;
            double d  = distance(&dx, &dy, &dz, particles[i], subParticles[j]);
            particles[i].f.x += dx * d;
            particles[i].f.y += dy * d;
            particles[i].f.z += dz * d;
        }//end-for (int j = 0; j < blockDim.x; j++){

    __syncthreads();
    }//end-for (int blk = 0; blk < gridDim.x; blk++){

    
}

__global__ void particleParticleVelocityPosition_k(tpParticle *particles, const double dt){
    int i  = blockDim.x * blockIdx.x + threadIdx.x;

    particles[i].v.x += dt * particles[i].f.x;
    particles[i].v.y += dt * particles[i].f.y;
    particles[i].v.z += dt * particles[i].f.z;

    particles[i].p.x += dt * particles[i].v.x;
    particles[i].p.y += dt * particles[i].v.y;
    particles[i].p.z += dt * particles[i].v.z;
}


void particleParticle (tpParticle *h_particles, int nParticles, int timesteps, double dt){
    int threadsF = BLOCK_SIZE_F,
        blocksF  = nParticles / BLOCK_SIZE_F,
        threadsVP = BLOCK_SIZE_VP,
        blocksVP  = nParticles / BLOCK_SIZE_VP;

    tpParticle *d_particles;


    if (nParticles < BLOCK_SIZE_F){
      blocksF = 1;
      threadsF = nParticles;
    }

    if (nParticles < BLOCK_SIZE_VP){
        blocksVP = 1;
        threadsVP = nParticles;
      }
  

    assert(hipDeviceReset()== hipSuccess);

    assert(hipMalloc((void**) &d_particles, nParticles * sizeof(tpParticle)) == hipSuccess);
    assert(hipMemcpy(d_particles, h_particles, nParticles * sizeof(tpParticle),  hipMemcpyHostToDevice) == hipSuccess);


    assert( ((nParticles % threadsF) == 0) && ((nParticles % threadsVP) == 0) );
    //fprintf(stdout, "\n B(%d) T(%d) \n", blocks, threads);
    //fprintf(stdout, "Shared memory allocated %d\n", threads * sizeof(tpParticle));
    

    for (int t = 0; t < timesteps; t++){
      //setup_kernel<<<blocos, threads,0, mStreams[i] >>>(time (NULL) + offset, mStates+offset);

      particleParticleForces_k<<<blocksF, threadsF, threadsF * sizeof(tpParticle)>>>(d_particles, dt);
      //assert( cudaDeviceSynchronize() == cudaSuccess);

      particleParticleVelocityPosition_k<<<blocksVP, threadsVP>>>(d_particles, dt);
      //assert( cudaDeviceSynchronize() == cudaSuccess);
    }//end-for (int t = 0; t < timesteps; t++){
    
    assert(hipMemcpy(h_particles, d_particles, nParticles * sizeof(tpParticle),  hipMemcpyDeviceToHost) == hipSuccess);
    hipFree(d_particles);
}

//--------------------------------------------------------------------------------------------------------
void printLog(tpParticle *particles, int nParticles, int timestep);
void initialCondition(tpParticle *particles, int nParticles);

int main (int ac, char **av){
    int timesteps  = atoi(av[1]),
        nParticles = atoi(av[2]),
        flagSave = atoi(av[3]);


    double              dt =  0.00001f;
    tpParticle *particles = NULL;


    fprintf(stdout, "\nParcile system particle to particle \n");
    fprintf(stdout, "Memory used %lu bytes \n", nParticles * sizeof(tpParticle));

    particles = (tpParticle *) malloc ( nParticles * sizeof(tpParticle));
    assert(particles != NULL);

    initialCondition(particles, nParticles);

    particleParticle(particles, nParticles, timesteps, dt);

    if (flagSave == 1)
      printLog(particles, nParticles, timesteps);
    free(particles);



}

void printLog(tpParticle *particles, int nParticles, int timestep){
    char fileName[128];
    sprintf(fileName, "%s-%d-log.bin", __FILE__,  timestep);
    fprintf(stdout, "Saving file [%s] ", fileName); fflush(stdout);
    FILE *ptr = fopen(fileName, "wb+");
    //fwrite ((const void*)particles , sizeof(tpParticle), nParticles, ptr);
    for(int i = 0; i < nParticles; i++)
        fprintf(ptr, "%d \t %.10f %.10f %.10f \t %.10f %.10f %.10f \t %.10f %.10f %.10f \n", i,  particles[i].p.x, particles[i].p.y, particles[i].p.z,  particles[i].v.x, particles[i].v.y, particles[i].v.z, particles[i].f.x, particles[i].f.y, particles[i].f.z);


    fclose(ptr);
    fprintf(stdout, "[OK]\n"); fflush(stdout);
}

void initialCondition(tpParticle *particles, int nParticles){

    srand(42);

    memset(particles, 0x00, nParticles * sizeof(tpParticle));

    for (int i = 0; i < nParticles ; i++){
        particles[i].p.x =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
        particles[i].p.y =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
        particles[i].p.z =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
      }




}
