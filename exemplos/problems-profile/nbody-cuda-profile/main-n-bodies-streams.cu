
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <string.h>
#define EPSILON 1E-9
#define BLOCK_SIZE_F 512 //Work with blocks of 512 threads due to double precision - shared memory
#define BLOCK_SIZE_VP 1024 //Work with blocks of 512 threads due to double precision - shared memory
//#define STREAMS        4
/*Declarando as structs de particula e forca*/
struct stCoord{
    double x,
          y,
          z;
};
typedef struct stCoord tpCoord;
struct stParticle
{
    tpCoord p,
            v,
            f;

};
typedef struct stParticle tpParticle;
//--------------------------------------------------------------------------------------------------------


__device__ double distance(  double* dx,
                            double* dy,
                            double* dz,
                            const tpParticle A,
                            const tpParticle B){
            double x = A.p.x - B.p.x;
            double y = A.p.y - B.p.y;
            double z = A.p.z - B.p.z;

            *dx = x; *dy = y; *dz = z;
            x *= x; y *= y; z *= z;
            return 1.0 / sqrt((double)x + y + z + EPSILON);

}


__global__ void particleParticleForces_k(tpParticle *p1, tpParticle *p2, const double dt, const int streams){
    extern __shared__ tpParticle subParticles[];
    int i  = blockDim.x * blockIdx.x + threadIdx.x;
    tpParticle p = p1[i];
//  __shared__ tpParticle subParticles[BLOCK_SIZE];
  
    for (int blk = 0; blk < gridDim.x * streams; blk++){
        subParticles[threadIdx.x] = p2[ blockDim.x * blk + threadIdx.x];
        __syncthreads();

        for (int j = 0; j < blockDim.x; j++){
            double dx = 0.0f,  dy = 0.0f, dz = 0.0f;
            double d  = distance(&dx, &dy, &dz, p, subParticles[j]);
            p.f.x += dx * d;
            p.f.y += dy * d;
            p.f.z += dz * d;
        }//end-for (int j = 0; j < blockDim.x; j++){

    __syncthreads();
    }//end-for (int blk = 0; blk < gridDim.x; blk++){

    p1[i] = p;
}

__global__ void particleParticleVelocityPosition_k(tpParticle *particles, const double dt){
    int i  = blockDim.x * blockIdx.x + threadIdx.x;
  
    particles[i].v.x += dt * particles[i].f.x;
    particles[i].v.y += dt * particles[i].f.y;
    particles[i].v.z += dt * particles[i].f.z;

    particles[i].p.x += dt * particles[i].v.x;
    particles[i].p.y += dt * particles[i].v.y;
    particles[i].p.z += dt * particles[i].v.z;
}


void particleParticle (hipStream_t  *streams , int nstreams, tpParticle *h_particles, int nParticles, int timesteps, double dt){
    int threadsF = 0,
        blocksF  = 0,
        threadsVP = 0,
        blocksVP  = 0,
        sPart = 0;

    tpParticle       *d_particles;
    
   

    sPart = nParticles / nstreams;


    threadsF  = BLOCK_SIZE_F;
    blocksF   = sPart / BLOCK_SIZE_F;
    threadsVP = BLOCK_SIZE_VP;
    blocksVP  = sPart / BLOCK_SIZE_VP;
    
    if (sPart < BLOCK_SIZE_F){
      blocksF = 1;
      threadsF = sPart;
    }

    if (sPart < BLOCK_SIZE_VP){
        blocksVP = 1;
        threadsVP = sPart;
      }
  

    
    assert(hipMalloc((void**) &d_particles, nParticles * sizeof(tpParticle)) == hipSuccess);
    assert(hipMemcpy(d_particles, h_particles, nParticles * sizeof(tpParticle),  hipMemcpyHostToDevice) == hipSuccess);

    

    assert( ((sPart % threadsF) == 0) && ((sPart % threadsVP) == 0) && ((nParticles % nstreams) == 0) );
    //fprintf(stdout, "\n B(%d) T(%d) \n", blocks, threads);
    //fprintf(stdout, "Shared memory allocated %d\n", threads * sizeof(tpParticle));
    

    for (int t = 0; t < timesteps; t++){
      
        for (int i = 0; i < nstreams; i++){
            int offset = sPart * i;
            particleParticleForces_k<<<blocksF, threadsF, threadsF * sizeof(tpParticle), streams[i]>>>(d_particles+offset, d_particles, dt, nstreams);
            particleParticleVelocityPosition_k<<<blocksVP, threadsVP,  0, streams[i]>>>(d_particles+offset, dt);
        }

/*
        for (int i = 0; i < STREAMS; i++){
            assert( cudaStreamSynchronize(streams[i]) == cudaSuccess );
        }

        for (int i = 0; i < STREAMS; i++){
            int offset = sPart * i;
            
        }
*/
        for (int i = 0; i < nstreams; i++){
            assert( hipStreamSynchronize(streams[i]) == hipSuccess );
        }
    }//end-for (int t = 0; t < timesteps; t++){
    
     

    assert(hipMemcpy(h_particles, d_particles, nParticles * sizeof(tpParticle),  hipMemcpyDeviceToHost) == hipSuccess);
    hipFree(d_particles);
}

//--------------------------------------------------------------------------------------------------------
void printLog(tpParticle *particles, int nParticles, int timestep);
void initialCondition(tpParticle *particles, int nParticles);

int main (int ac, char **av){
    int timesteps  = atoi(av[1]),
        nParticles = atoi(av[2]),
        flagSave = atoi(av[3]),
        nstreams = atoi(av[4]);

    hipStream_t  *streams       =  (hipStream_t *) malloc (nstreams * sizeof(hipStream_t));
    double         dt            =  0.00001f;
    tpParticle    *h_Particles   = NULL;
   

    fprintf(stdout, "\nParcile system particle to particle \n");
    fprintf(stdout, "Memory used %lu bytes \n", nParticles * sizeof(tpParticle));

    assert(hipDeviceReset()== hipSuccess);
    assert(hipHostMalloc((void**) (&h_Particles), nParticles * sizeof(tpParticle), hipHostMallocDefault) == hipSuccess);
    assert(h_Particles != NULL);
    
   
    for (int i = 0; i < nstreams; i++)
       assert(hipStreamCreate(&streams[i]) == hipSuccess);

    initialCondition(h_Particles, nParticles);

    particleParticle(streams, nstreams, h_Particles, nParticles, timesteps, dt);

    if (flagSave == 1)
      printLog(h_Particles, nParticles, timesteps);

    for (int i = 0; i < nstreams; i++)
      assert(hipStreamDestroy(streams[i]) == hipSuccess); 
  
    hipHostFree(h_Particles);   
    



}

void printLog(tpParticle *particles, int nParticles, int timestep){
    char fileName[128];
    sprintf(fileName, "%s-%d-log.bin", __FILE__,  timestep);
    fprintf(stdout, "Saving file [%s] ", fileName); fflush(stdout);
    FILE *ptr = fopen(fileName, "wb+");
    //fwrite ((const void*)particles , sizeof(tpParticle), nParticles, ptr);
    for(int i = 0; i < nParticles; i++)
        fprintf(ptr, "%d \t %.10f %.10f %.10f \t %.10f %.10f %.10f \t %.10f %.10f %.10f \n", i,  particles[i].p.x, particles[i].p.y, particles[i].p.z,  particles[i].v.x, particles[i].v.y, particles[i].v.z, particles[i].f.x, particles[i].f.y, particles[i].f.z);


    fclose(ptr);
    fprintf(stdout, "[OK]\n"); fflush(stdout);
}

void initialCondition(tpParticle *particles, int nParticles){

    srand(42);

    memset(particles, 0x00, nParticles * sizeof(tpParticle));

    for (int i = 0; i < nParticles ; i++){
        particles[i].p.x =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
        particles[i].p.y =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
        particles[i].p.z =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
      }




}
