#include <iostream>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
using namespace std;
__global__ void AddVet(int *c, int *a, int *b){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = a[idx] + b[idx];
    
}
int main (int argc, char **argv){
    //C = A + B -> Objetivo do código é somar vetores de inteiros
    
    int h_vetS = 16;
    
    int *h_A = NULL;
    int *h_B = NULL;
    int *h_C = NULL;
    
    int *d_A = NULL;
    int *d_B = NULL;
    int *d_C = NULL;
    
    
    h_A = new int [h_vetS];
    h_B = new int [h_vetS];
    h_C = new int [h_vetS]; 
    
    hipDeviceReset();
    
    hipMalloc((void**)&d_A, sizeof(int) * h_vetS);
    hipMalloc((void**)&d_B, sizeof(int) * h_vetS);
    hipMalloc((void**)&d_C, sizeof(int) * h_vetS);
    
    //Inicializando as variáveis
    for (int  i = 0 ; i < h_vetS; i++){
        h_A[i] = i + 1;
        h_B[i] = (i + 1) * 10;
        h_C[i] = 0;
    }
    
    //Cópia host -> device
    hipMemcpy(d_A, h_A, sizeof(int) * h_vetS, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(int) * h_vetS, hipMemcpyHostToDevice);
    
    
    
    AddVet<<<2, 8>>>(d_C, d_A, d_B);
    
    //Código sequencial
/*
    for (int  i = 0 ; i < h_vetS; i++){
        h_C[i] = h_A[i] + h_B[i];
        ~
    }
  */  
    //Cópia device -> host
    hipMemcpy(h_C, d_C, sizeof(int) * h_vetS, hipMemcpyDeviceToHost);
    
    //Exibir resultado
    for (int  i = 0 ; i < h_vetS; i++){
        cout << "i = " << i << " h_C[i] = " << h_C[i] << endl;
        
    }
    
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    
    return EXIT_SUCCESS;
}
