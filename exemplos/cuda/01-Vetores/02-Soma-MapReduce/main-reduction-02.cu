#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <cstdio>
#include "App.h"

#define CHECK_ERROR(call) do {                                                    \
   if( hipSuccess != call) {                                                             \
      std::cerr << std::endl << "CUDA ERRO: " <<                             \
         hipGetErrorString(call) <<  " in file: " << __FILE__                \
         << " in line: " << __LINE__ << std::endl;                               \
         exit(0);                                                                                 \
   } } while (0)


using namespace std;

__global__ void Soma(float *vet, const int offset){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int a = offset*i;
	int b = a+(offset/2);
	float v = vet[a] + vet[b];
	vet[a] = v;



}
__global__  void Soma2(float *b, float *a){

   extern __shared__ float partialSum[];

    int t = threadIdx.x;

   partialSum[t] = a[t];
   __syncthreads();


   for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
      __syncthreads();
      if (t % (2* (stride)) == 0)
        partialSum[t] += partialSum[t+stride];
   }

   __syncthreads();
   if (threadIdx.x  == 0)
      b[t] = partialSum[t];


}

__global__  void Soma1(float *b, float *a){

   extern __shared__ float partialSum[];

   int t = threadIdx.x;

   partialSum[t] = a[t];
   __syncthreads();

   for (unsigned int stride = 1; stride < blockDim.x; stride *=2){
      __syncthreads();
      if (t % (2* (stride)) == 0)
        partialSum[t] += partialSum[t+stride];
   }

   __syncthreads();
   if (threadIdx.x  == 0)
      b[t] = partialSum[t];


}

int main (int argc, char **argv){
   int h_Size = 16;

   float   *h_VetA = NULL,
           *d_VetA = NULL;


   Stopwatch sw;

   size_t free = 0,
          total = 0;
   cout << endl << "CUDA runtime versao: " << CUDART_VERSION << endl;

   //Reset no device
   CHECK_ERROR(hipDeviceReset());

   //Verificando espaço livre em memória
   CHECK_ERROR(hipMemGetInfo(&free, &total));
   cout << "Memoria livre: " << (free / 1024 / 1024)   << " MB\n";
   cout << "Memoria total: " << (total / 1024 / 1024)  << " MB\n";




   //Aloca memória GPU
   CHECK_ERROR(hipMalloc((void**) &d_VetA, h_Size * sizeof(float)));



   //Alocando memória na CPU
   h_VetA = new float [h_Size];


   //Preenchendo vetores
   for (int i = 0; i < h_Size; i++){
      h_VetA[i] = static_cast <float> (i+1);
      cout << h_VetA[i] << " ";
   }
   cout << endl;

   FREQUENCY(sw);

   START_STOPWATCH(sw);
   //Copiando CPU --> GPU
   CHECK_ERROR(hipMemcpy(d_VetA, h_VetA, h_Size * sizeof(float),  hipMemcpyHostToDevice));


   int numBlocks = 1;
   int threadsPerBlock = 4;
/*
   h_Size = 131072;
   int j = 1024;
   int i = 2;
   int l = 1;
      for (int k = 32; k > 0; k >>=1){
		   for (int j = 0; j < k; j++)
			   cout << i*j << ", " << i*j+l << endl;
	   i <<=1;
	   l <<= 1;
	   cout << "----" << endl;
	   cin.get();
      }


   exit(-1);
*/


	cout << "Blocos/threads : " << numBlocks << "," << threadsPerBlock << endl;

	int offSet = 2;
	for (int i = h_Size/2; i > 0; i>>=1){
		threadsPerBlock = i;
		Soma <<< numBlocks, threadsPerBlock >>> (d_VetA, offSet);
		offSet <<= 1;
		CHECK_ERROR(hipDeviceSynchronize());
		printf("\n");


	}



   /*
   //for (int i = 0; i < 2 ; i++){
	  Soma1<<<numBlocks, threadsPerBlock,  threadsPerBlock * sizeof(float) >>> (d_VetB, d_VetA);
	  CHECK_ERROR(hipDeviceSynchronize());
	 //
  // }
*/

   CHECK_ERROR(hipMemcpy(h_VetA, d_VetA, h_Size * sizeof(float),  hipMemcpyDeviceToHost));
   STOP_STOPWATCH(sw);

   cout << endl << "Tempo gasto [GPU+MEM]: " << sw.mElapsedTime << " (ms)" << endl;

   cout <<  endl << "Resultado: "<<  endl;

/*
   for (int i = 0; i < h_Size; i++)
      cout << h_VetA[i] << endl;
*/
      cout << h_VetA[0] << endl;


   CHECK_ERROR(hipFree(d_VetA));  //Liberando memorias GPU e CPU

   delete[] h_VetA;


   cout << "FIM" << endl;

   return EXIT_SUCCESS;
}
