#include "hip/hip_runtime.h"
/*
 *************************************************************************
   unsigned int width = gridDim.x * blockDim.x;
   unsigned int height = gridDim.y * blockDim.y;
   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
   unsigned int kn = y * width + x;
 *************************************************************************
 N blocks x M threads  <---- IPC
*/

#include <cstdio>
#include <GPU.h>
#include <cv.h>
#include <highgui.h>
#include <App.h>
#include <cassert>
#include <iostream>
//#define BSIZE_X 24
//#define BSIZE_Y 8

#define BSIZE_X 12
#define BSIZE_Y 4

using namespace std;
__global__
void kernel_Sobel_S2(uchar *output, float *input, const int w, const int h, const float threshold){
//void kernel_Sobel_G(uchar *output, float *input, const int width, const int height, const float threshold){


	__shared__ float sharedMem[BSIZE_Y+(2*1)][BSIZE_X+(2*1)];

	    int tx     = threadIdx.x,
	        ty     = threadIdx.y,
	        width  = gridDim.x * blockDim.x,
	        height = gridDim.y * blockDim.y,
	        x      = blockDim.x * blockIdx.x + threadIdx.x,
	        y      = blockDim.y * blockIdx.y + threadIdx.y,
	        kn     = y * width + x,
	        sX     = tx + 1,
	        sY     = ty + 1;





	   sharedMem[sY][sX] =  input[kn] ;

	   if (threadIdx.x == 0)
	   {
	      if (blockIdx.x == 0){
		       sharedMem[sY][sX-1] = 0.0f;
		       sharedMem[sY-1][sX-1] = 0.0f;
             sharedMem[sY][sX+blockDim.x] = input[kn+blockDim.x] ;

	      }else if (blockIdx.x  == (gridDim.x - 1)){
		       sharedMem[sY][sX-1] = input[kn-1] ;
		       sharedMem[sY][sX+blockDim.x] = 0.0f;


         }else{
		       sharedMem[sY][sX-1] = input[kn-1] ;
		       sharedMem[sY][sX+blockDim.x] = input[kn+blockDim.x] ;

		       sharedMem[sY+1][sX-1] = input[kn-1 + width] ;
		       sharedMem[sY+1][sX+blockDim.x] = input[kn+blockDim.x + width] ;

		       sharedMem[sY-1][sX-1] = input[kn-1-width] ;
		       sharedMem[sY-1][sX+blockDim.x] = input[kn+blockDim.x-width] ;
               

         }

	   }

	   if (threadIdx.y == 0)
	   {
	      if (blockIdx.y == 0){
		       sharedMem[sY-1][sX] = 0.0f;
		       sharedMem[sY+blockDim.y][sX] = input[kn+(blockDim.y*width)] ;
            
	      }else if (blockIdx.y  == (gridDim.y - 1)){
		       sharedMem[sY-1][sX] = input[kn-width] ;
		       sharedMem[sY+blockDim.y][sX] = 0.0f;

         }else{
		       sharedMem[sY-1][sX] = input[kn-width] ;
		       sharedMem[sY+blockDim.y][sX] = input[kn+(blockDim.y * width)] ;
  
         }

	   }


	   __syncthreads();





	float sum_X = sharedMem[sY+1][sX-1] +       \
		        2.0f * sharedMem[sY+1][sX]  +   \
		        sharedMem[sY+1][sX+1]       -   \
		        sharedMem[sY-1][sX-1]       -   \
		        2.0f * sharedMem[sY-1][sX]  -   \
		        sharedMem[sY-1][sX+1];

	float sum_Y = sharedMem[sY-1][sX+1]  +       \
				2.0f * sharedMem[sY][sX+1] +    \
				sharedMem[sY+1][sX+1]  -         \
				sharedMem[sY-1][sX-1] -       \
				2.0f * sharedMem[sY][sX-1] -  \
				sharedMem[sY+1][sX-1];



	float xy = sqrt(fabs(sum_X) + fabs(sum_Y));

	if (xy > threshold)
		output[kn] = 255;// static_cast <uchar> (xy);;
	else
		output[kn] = 0;



}


//-------------
__global__
void kernel_Sobel_S(uchar *output, float *input, const int w, const int h, const float threshold){
//void kernel_Sobel_G(uchar *output, float *input, const int width, const int height, const float threshold){


	__shared__ float sharedMem[BSIZE_Y+(2*1)][BSIZE_X+(2*1)];

	    int tx     = threadIdx.x,
	        ty     = threadIdx.y,
	        width  = gridDim.x * blockDim.x,
	        height = gridDim.y * blockDim.y,
	        x      = blockDim.x * blockIdx.x + threadIdx.x,
	        y      = blockDim.y * blockIdx.y + threadIdx.y,
	        kn     = y * width + x,
	        sX     = tx + 1,
	        sY     = ty + 1;





	   sharedMem[sY][sX] =  input[kn] ;

	   if (threadIdx.x == 0)
	   {
	      if (blockIdx.x == 0){
		       sharedMem[sY][sX-1] = 0.0f;
		       sharedMem[sY-1][sX-1] = 0.0f;
             sharedMem[sY][sX+blockDim.x] = input[kn+blockDim.x] ;

	      }else if (blockIdx.x  == (gridDim.x - 1)){
		       sharedMem[sY][sX-1] = input[kn-1] ;
		       sharedMem[sY][sX+blockDim.x] = 0.0f;


         }else{
		       sharedMem[sY][sX-1] = input[kn-1] ;
		       sharedMem[sY][sX+blockDim.x] = input[kn+blockDim.x] ;

		       sharedMem[sY+1][sX-1] = input[kn-1 + width] ;
		       sharedMem[sY+1][sX+blockDim.x] = input[kn+blockDim.x + width] ;

		       sharedMem[sY-1][sX-1] = input[kn-1-width] ;
		       sharedMem[sY-1][sX+blockDim.x] = input[kn+blockDim.x-width] ;
               

         }

	   }

	   if (threadIdx.y == 0)
	   {
	      if (blockIdx.y == 0){
		       sharedMem[sY-1][sX] = 0.0f;
		       sharedMem[sY+blockDim.y][sX] = input[kn+(blockDim.y*width)] ;
            
	      }else if (blockIdx.y  == (gridDim.y - 1)){
		       sharedMem[sY-1][sX] = input[kn-width] ;
		       sharedMem[sY+blockDim.y][sX] = 0.0f;

         }else{
		       sharedMem[sY-1][sX] = input[kn-width] ;
		       sharedMem[sY+blockDim.y][sX] = input[kn+(blockDim.y * width)] ;
  
         }

	   }


	   __syncthreads();





	float sum_X = sharedMem[sY+1][sX-1] +       \
		        2.0f * sharedMem[sY+1][sX]  +   \
		        sharedMem[sY+1][sX+1]       -   \
		        sharedMem[sY-1][sX-1]       -   \
		        2.0f * sharedMem[sY-1][sX]  -   \
		        sharedMem[sY-1][sX+1];

	float sum_Y = sharedMem[sY-1][sX+1]  +       \
				2.0f * sharedMem[sY][sX+1] +    \
				sharedMem[sY+1][sX+1]  -         \
				sharedMem[sY-1][sX-1] -       \
				2.0f * sharedMem[sY][sX-1] -  \
				sharedMem[sY+1][sX-1];



	float xy = sqrt(fabs(sum_X) + fabs(sum_Y));

	if (xy > threshold)
		output[kn] = 255;// static_cast <uchar> (xy);;
	else
		output[kn] = 0;



}
 

__global__
void kernel_Sobel_G(uchar *output, float *input, const int width, const int height, const float threshold){
	unsigned int x      = blockDim.x * blockIdx.x + threadIdx.x,
			     y      = blockDim.y * blockIdx.y + threadIdx.y;

	float sum_X = 0.0f,
		  sum_Y = 0.0f;

	if (x == 0 || x == width - 1 || y == 0 || y == height - 1)
		output[y * width + x] = 0;
	else{
		sum_X = input[(y+1) * width + (x-1)] +       \
		        2.0f * input[(y+1) * width + (x)] +  \
		        input[(y+1) * width + (x+1)] -       \
		        input[(y-1) * width + (x-1)] -       \
		        2.0f * input[(y-1) * width + (x)] -  \
		        input[(y-1) * width + (x+1)];

	    sum_Y = input[(y-1) * width + (x+1)] +       \
				2.0f * input[(y) * width + (x+1)] +  \
				input[(y+1) * width + (x+1)] -       \
				input[(y-1) * width + (x-1)] -       \
				2.0f * input[(y) * width + (x-1)] -  \
				input[(y+1) * width + (x-1)];
	 }


	float xy = sqrt(fabs(sum_X) + fabs(sum_Y));

	if (xy > threshold)
		output[y * width + x] = 255;// static_cast <uchar> (xy);;
	else
		output[y * width + x] = 0;



}
__global__
void kernel_RGB2YIQ(float *output, uchar *input, const int width){
	unsigned int x      = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y      = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int ptrIn   = (y * width * 3) + (x * 3);
	unsigned int ptrOut  = y * width  + x ;

	float fR = 0.0f, fG = 0.0f, fB = 0.0f,
			fY = 0.0f;//, fI = 0.0f, fQ = 0.0f;

	fR = static_cast<float> (input[ptrIn+0]) / 255.0f;
	fG = static_cast<float> (input[ptrIn+1]) / 255.0f;
	fB = static_cast<float> (input[ptrIn+2]) / 255.0f;

	fY = 0.299f    * fR + 0.587f     * fG +  0.114f    * fB;
	//fI = 0.595716f * fR + -0.274453f * fG + -0.321263f * fB;
	//fQ = 0.211456f * fR + -0.522591f * fG +  0.311135f * fB;
	output[ptrOut] =  fY * 255.0f;

}

IplImage * GPU_RGB2YIQ(IplImage *inputImg, const float threshold, char type) {
// get the image data
  int height    = inputImg->height,
	  width     = inputImg->width,
	  imgSize   =  width * height;

  IplImage *newImg = cvCreateImage(cvSize(width,height),IPL_DEPTH_8U,1); //Apenas tons de cinza

  uchar *srcImg = NULL,
		*sblImg = NULL;

  float *iyqImg = NULL;


  dim3 blocks,  //= dim3(width/BSIZE_X, height/BSIZE_Y, 1),
	   threads;// = dim3(BSIZE_X, BSIZE_Y, 1);

  blocks.x = width/BSIZE_X;
  blocks.y = height/BSIZE_Y;
  blocks.z = 1;

  threads.x = BSIZE_X;
  threads.y = BSIZE_Y;
  threads.z = 1;

  assert(threads.x * threads.y <= 512);

   cout << "Threads (" << threads.x << "," << threads.y << ")" << endl;
   cout << "Blocks  (" << blocks.x << "," << blocks.y << ")" << endl;


  CHECK_ERROR(hipMalloc((void**) &srcImg, imgSize * 3));
  CHECK_ERROR(hipMalloc((void**) &iyqImg, imgSize * sizeof(float)));
  CHECK_ERROR(hipMalloc((void**) &sblImg, imgSize));

  CHECK_ERROR(hipMemcpy(srcImg, reinterpret_cast <uchar *> (inputImg->imageData), imgSize * 3, hipMemcpyHostToDevice));

  Stopwatch stopwatch;
  FREQUENCY(stopwatch);

  START_STOPWATCH(stopwatch);
  kernel_RGB2YIQ<<<blocks, threads>>> (iyqImg, srcImg,  width);
  //CHECK_ERROR(hipDeviceSynchronize());
  if (type == 'G'){
      kernel_Sobel_G<<<blocks, threads>>> (sblImg, iyqImg,  width, height, threshold);
    //  cout << "GLOBAL" << endl;  
   }else{
      kernel_Sobel_S2<<<blocks, threads>>> (sblImg, iyqImg,  width, height, threshold);
  //    cout << "SHARED" << endl;
   }
  CHECK_ERROR(hipDeviceSynchronize());
  STOP_STOPWATCH(stopwatch);
  cout << "Elapsed time: " << stopwatch.mElapsedTime << "ms" << endl;


  CHECK_ERROR(hipMemcpy(reinterpret_cast <uchar *> (newImg->imageData), sblImg, imgSize, hipMemcpyDeviceToHost));

  return newImg;
}



//



