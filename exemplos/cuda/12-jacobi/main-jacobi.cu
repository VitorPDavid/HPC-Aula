#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#define ERROR 1E-16
#define EPSILON 1E-17
#define MAXTHREADS 1024

struct stMatrix{
    double *A;
    double *B;
    double *X1;
    double *X0;
    int mn;
};
typedef struct stMatrix tpMatrix;
//------------------------------------------------------------------------------------------
/*
 * Kernel sem memória compartilhada para resolver sistema linear de jacobi
 */
 __global__ void JacobiMethodKernel_B( double *__restrict__ A,
                                       double *__restrict__ X1,
                                       double *__restrict__ X0,
                                       double *__restrict__ B,
                                       double *__restrict__ err,
                                       const unsigned int mn){
  
  extern __shared__ double buffer[];                                     
  unsigned int idx  =  (blockIdx.x * blockDim.x) + threadIdx.x;
  double  aux    = 0.0, 
          div    = 0.0f;

  for (unsigned b = 0; b < gridDim.x; b++){
    buffer[threadIdx.x] = X0[(b * blockDim.x) + threadIdx.x];
    __syncthreads();
    
    for (unsigned int i = 0; i < blockDim.x; i++){  
        
      unsigned int j = (b * blockDim.x) + i;
        
        if (idx != j)
          aux += (A[mn*idx + j] * buffer[i]);
        else
          div =  A[mn*idx + j];
    }//end-for (unsigned int i = 0; i < blockDim.x; i++){  

    __syncthreads();
  }//end-for (unsigned b = 0; b < gridDim.x; b++){
  


  X1[idx] =  ((B[idx] - aux) / div );
  err[idx] =  fabs((X1[idx] - X0[idx]) / (X1[idx] + EPSILON));
}
//------------------------------------------------------------------------------------------
/*
 * Kernel sem memória compartilhada para resolver sistema linear de jacobi
 */
__global__ void JacobiMethodKernel_A( double *__restrict__ A,
                                      double *__restrict__ X1,
                                      double *__restrict__ X0,
                                      double *__restrict__ B,
                                      double *__restrict__ err,
                                      const unsigned int mn){
  unsigned int idx  =  (blockIdx.x * blockDim.x) + threadIdx.x;
  double aux = 0.0, 
  valueX = 0.0, 
  div = 0.0f;
  for (unsigned int i = 0; i < mn; i++){
    valueX = X0[i];
    if (idx != i)
      aux += (A[mn*idx + i] * valueX);
    else
      div =  A[mn*idx + i];
  }
  X1[idx] =  ((B[idx] - aux) / div );
  err[idx] =  fabs((X1[idx] - X0[idx]) / (X1[idx] + EPSILON));
}


//------------------------------------------------------------------------------------------
/*
 * Kernel que acha a max do conjunto - usa apenas 1 bloco
 */

__global__  void findMax(double *X, const unsigned int blocks){
  extern __shared__ double buffer[];
  unsigned int i = threadIdx.x;
  buffer[threadIdx.x] = X[i];

  for (unsigned int j = 0; j < blocks; j++){
    buffer[threadIdx.x] = max(buffer[threadIdx.x], X[threadIdx.x + j * blockDim.x]);
  }
//Se deslocar para os outros blocosk

  
  for (int k = (blockDim.x >> 1); k >= 1; k >>= 1){
      __syncthreads();
      if (threadIdx.x < k){
        buffer[threadIdx.x] = max(buffer[threadIdx.x], buffer[threadIdx.x + k]);
      }
  }
  
  __syncthreads();
  if (threadIdx.x == 0)
    X[threadIdx.x] = buffer[threadIdx.x];


}


//------------------------------------------------------------------------------------------
void LoadMatrixAndVector(char *matrixFile, char *vectorFile, tpMatrix *matrix);
void PrintMatrixAndVector(const tpMatrix *matrix);
void JacobiMethodGPU(tpMatrix *matrix);
void JacobiMethodCPU(tpMatrix *matrix);
void PrintX(double *X, const int size);
int main(int ac, char**av) {
  tpMatrix matrix;
  
  int      flagSave = atoi(av[4]);
  fprintf(stdout, "\nMétodo iterativo de solução de sistema linear - Jacobi\n");
  
  matrix.mn = atoi(av[3]);
  matrix.A = (double*) malloc (matrix.mn * matrix.mn *  sizeof(double));
  matrix.B = (double*) malloc (matrix.mn * sizeof(double));
  matrix.X0 = (double*) malloc (matrix.mn *  sizeof(double));
  matrix.X1 = (double*) malloc (matrix.mn *  sizeof(double));
  memset(matrix.X0, 0x00, matrix.mn *  sizeof(double));
  memset(matrix.X1, 0x00, matrix.mn *  sizeof(double));

  LoadMatrixAndVector(av[1], av[2], &matrix);
//    PrintMatrixAndVector(&matrix);
  //JacobiMethod(&matrix, iter);
  JacobiMethodGPU(&matrix);

  if (flagSave == 1)
    PrintX(matrix.X0, matrix.mn);



  free(matrix.A);
  free(matrix.B);
  free(matrix.X0);

  return EXIT_SUCCESS;
}

/*
 * Carrega a matrix e o vetor B do arquivo
 */
void LoadMatrixAndVector(char *matrixFile, char *vectorFile, tpMatrix *matrix){
  FILE *ptr = fopen(matrixFile, "rb+");
  assert(ptr != NULL);
  fread (matrix->A,sizeof(double), matrix->mn * matrix->mn, ptr);
  fclose(ptr);

  ptr = fopen(vectorFile, "rb+");
  assert(ptr != NULL);
  fread (matrix->B,sizeof(double), matrix->mn , ptr);
    
  fclose(ptr);
}

void PrintMatrixAndVector(const tpMatrix *matrix){

  fprintf(stdout, "Matrix (%d, %d)\n", matrix->mn, matrix->mn);
  for (int j = 0; j < matrix->mn; j++){
    for (int i = 0; i < matrix->mn; i++){
      int k = j * matrix->mn + i;
      fprintf(stdout, "%.7f ", matrix->A[k]);
    }
   fprintf(stdout, " \t %.7f \n", matrix->B[j]);
  }
}

void JacobiMethodGPU(tpMatrix *matrix){
  unsigned int blocks = 0,
               threads = MAXTHREADS;

  double *d_A  = NULL, 
         *d_B  = NULL, 
         *d_X0 = NULL, 
         *d_X1 = NULL, 
         *d_Err = NULL,
         h_err = 0.0;

  int inter = 0;       

  unsigned int memCudaDataMatrixSize = sizeof(double) * (matrix->mn * matrix->mn);
  unsigned int memCudaDataVectorSize = sizeof(double) * (matrix->mn);
  assert(hipMalloc((void**) &d_A, memCudaDataMatrixSize) == hipSuccess);
  assert(hipMalloc((void**) &d_X0, memCudaDataVectorSize) == hipSuccess);
  assert(hipMalloc((void**) &d_X1, memCudaDataVectorSize) == hipSuccess);
  assert(hipMalloc((void**) &d_B, memCudaDataVectorSize) == hipSuccess);         
  assert(hipMalloc((void**) &d_Err, memCudaDataVectorSize) == hipSuccess);         

  assert(hipMemcpy(d_A, matrix->A, memCudaDataMatrixSize, hipMemcpyHostToDevice) == hipSuccess);
  assert(hipMemcpy(d_B, matrix->B, memCudaDataVectorSize, hipMemcpyHostToDevice) == hipSuccess);

  assert(hipMemset(d_X0, 0x00, memCudaDataVectorSize) == hipSuccess);
  assert(hipMemset(d_X1, 0x00, memCudaDataVectorSize) == hipSuccess);

  if (matrix->mn < MAXTHREADS){
    blocks = 1;
    threads = matrix->mn;
  }else{
    blocks = matrix->mn / MAXTHREADS;
  }

  do{
        
        //JacobiMethodKernel_A<<<blocks, threads >>> (d_A, d_X1, d_X0, d_B, d_Err, matrix->mn);       //Here, the kernel is called  by the main program!
        JacobiMethodKernel_B<<<blocks, threads, threads * sizeof(double) >>> (d_A, d_X1, d_X0, d_B, d_Err, matrix->mn);       //Here, the kernel is called  by the main program!
        assert(hipDeviceSynchronize() == hipSuccess);

        findMax<<<1, threads, threads * sizeof(double)>>> (d_Err, blocks);       //Here, the kernel is called  by the main program!
        assert(hipDeviceSynchronize() == hipSuccess);

        assert(hipMemcpy(&h_err, d_Err, sizeof(double), hipMemcpyDeviceToHost) == hipSuccess);

        double *swap = d_X0;
        d_X0 = d_X1;
        d_X1 = swap; 
        inter++;

  }while (h_err > ERROR);
  printf("\n%18.16lf %d\n", h_err, inter);

  assert(hipMemcpy(matrix->X0, d_X0, memCudaDataVectorSize, hipMemcpyDeviceToHost) == hipSuccess);

  hipFree(d_A);
  hipFree(d_X1);
  hipFree(d_X0);
  hipFree(d_B);
  hipFree(d_Err);



}

/*
 *
 */
void JacobiMethodCPU(tpMatrix *matrix){

   double aux, 
         div, 
         err = 0.0;
   int inter = 0;
   
   do{
         for (int j = 0; j < matrix->mn; j++){
            aux    = 0.0;
            div    = 0.0;
            
            for (int i = 0; i < matrix->mn; i++){
              if (j != i)
//                  aux += ((matrix->A[matrix->mn * j + i] * matrix->X0[i]) * -1.0);
                  aux += (matrix->A[matrix->mn * j + i] * matrix->X0[i]);
               else
                  div =  matrix->A[matrix->mn * j + i];
            }
            matrix->X1[j] = ((matrix->B[j] - aux) / div );
         }

         err = fabs((matrix->X1[0] - matrix->X0[0]) / (matrix->X1[0] + EPSILON));
         for (int j = 1; j < matrix->mn; j++){
            double b = fabs((matrix->X1[j] - matrix->X0[j]) / (matrix->X1[j] + EPSILON));
             err = max(err, b);   
         }

         double *swap = matrix->X0;
         matrix->X0 = matrix->X1;
         matrix->X1 = swap; 
         inter++;
   
       
   }while (err > ERROR);
   printf("\n%18.16lf %d\n", err, inter);

}
/*
 *
 */
void PrintX(double *X, const int size){
  FILE *ptr = fopen("solucao.txt", "w+");
  assert(ptr != NULL);
  for (int i = 0; i < size; i++){
    fprintf(ptr, "%20.15lf ", X[i]);
  }
  fprintf(ptr, "\n");
  fclose(ptr);
}


