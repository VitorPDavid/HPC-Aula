///usr/local/cuda/bin/nvprof -u s --print-gpu-summary ./gol-cuda 1024 1024 1000 0 0.1

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <assert.h>
#include <omp.h>
struct stLattice{
    unsigned char *buff0;
    unsigned char *buff1;
    int width;
    int height;
    int steps;
};
typedef struct stLattice tpLattice;


__global__
void kernel( unsigned char *buff1, unsigned char  *buff0){
    unsigned int x      = blockDim.x * blockIdx.x + threadIdx.x,
                 y      = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned char  cell = 0;
    unsigned char  myCell = buff0[y * blockDim.x * gridDim.x + x] ;
    unsigned char  newC = 0;
    //buff1[y * blockDim.x * gridDim.x + x] = 1 - buff0[y * blockDim.x * gridDim.x + x];

    if ((x > 0) && ( x < ((blockDim.x * gridDim.x) - 1)) && (y > 0) && (y < ((blockDim.y * gridDim.y) - 1))){
        cell = buff0[(y+1) * blockDim.x * gridDim.x + (x-1)] +  \
               buff0[(y+1) * blockDim.x * gridDim.x + (x)]   +  \
               buff0[(y+1) * blockDim.x * gridDim.x + (x+1)] +  \
               buff0[(y-1) * blockDim.x * gridDim.x + (x-1)] +  \
               buff0[(y-1) * blockDim.x * gridDim.x + (x)]   +  \
               buff0[(y-1) * blockDim.x * gridDim.x + (x+1)] +  \
               buff0[y * blockDim.x * gridDim.x + (x-1)] +  \
               buff0[y * blockDim.x * gridDim.x + (x+1)] ;

    }


    if ((cell == 3) && (myCell == 0))
        newC = 1;

    if ((cell >= 2) && (cell <= 3) && (myCell == 1))
        newC = 1;


    buff1[y  * blockDim.x * gridDim.x  +  x] = newC;
}

void GameOfLifeGPU(tpLattice *mLattice);
void InitRandnessGPU(tpLattice *mLattice, float p);
void print2File(tpLattice *mLattice);
int main(int ac, char**av)
{
    tpLattice mLattice;
    int flagSave = atoi(av[4]);
    float prob   = atof(av[5]);
    //Inicializa variável
    mLattice.width  = atoi(av[1]);
    mLattice.height = atoi(av[2]);
    mLattice.steps  = atoi(av[3]);

    fprintf(stdout, "\nGame of life");
    fprintf(stdout, "\nDominio(%d, %d, %d) Prob. %5.3f\n",   mLattice.width,   mLattice.height, mLattice.steps, prob);

    fflush(stdout);

    assert(hipDeviceReset() == hipSuccess);

    assert(hipMalloc((void**) &mLattice.buff0, mLattice.width *   mLattice.height *  sizeof(unsigned char)) == hipSuccess);
    assert(hipMalloc((void**) &mLattice.buff1, mLattice.width *   mLattice.height *  sizeof(unsigned char)) == hipSuccess);

//    mLattice.buff0 = (unsigned char*) malloc (mLattice.width *   mLattice.height *  sizeof(unsigned char));
//    mLattice.buff1 = (unsigned char*) malloc (mLattice.width *   mLattice.height *  sizeof(unsigned char));
    InitRandnessGPU(&mLattice, prob);
    GameOfLifeGPU(&mLattice);

    if (flagSave == 1)
      print2File(&mLattice);

    hipFree(mLattice.buff0);
    hipFree(mLattice.buff1);
    return EXIT_SUCCESS;
}

/*
 * Função utilizada para iniciar a matriz. Não mudar o valor constante do seed do rand
 */
void InitRandnessGPU(tpLattice *mLattice, float p){
  hipMemset(mLattice->buff0, 0x00,  mLattice->width *   mLattice->height *  sizeof(unsigned char));
  hipMemset(mLattice->buff1, 0x00,  mLattice->width *   mLattice->height *  sizeof(unsigned char));
  unsigned char *buff = (unsigned char *)malloc(mLattice->width *   mLattice->height *  sizeof(unsigned char));
  bzero(buff, mLattice->width *   mLattice->height *  sizeof(unsigned char));
  srand (42);
  for (int j = 1; j < mLattice->height - 1; j++){
      for (int i = 1; i < mLattice->width - 1; i++){
          int k = j * mLattice->width  +  i;
          float r = (rand() / (float)RAND_MAX);
          if (r <= p)
            buff[k] = 1;

      }//end-  for (int i = 0; i < mLattice->width; i++){
  }//end-for (int j = 0; j < mLattice->height; j++){

  assert(hipMemcpy(mLattice->buff0, buff, mLattice->width *   mLattice->height *  sizeof(unsigned char),  hipMemcpyHostToDevice) == hipSuccess);
  free(buff);

}//end-void InitRandness(tpLattice *mLattice, float p){

/*
 * Função que resolve o GOL chamando o kernel da GPU
 */
 void GameOfLifeGPU(tpLattice *mLattice){
     dim3 threads = dim3(32, 32, 1);
     dim3 blocks  = dim3(mLattice->width / threads.x,  mLattice->height / threads.y, 1);

     for (int i = 0; i < mLattice->steps; i++){
       kernel<<<blocks, threads>>> (mLattice->buff1, mLattice->buff0);
       assert(hipDeviceSynchronize() == hipSuccess);
       unsigned char *swap = mLattice->buff0;
       mLattice->buff0 = mLattice->buff1;
       mLattice->buff1 = swap;

     }


 }

/*
 * Função para imprimir para arquivo. Formato do arquivo .txt
 */
void print2File(tpLattice *mLattice)
{
  fprintf(stdout, "Save to file: game_of_life.txt");
  FILE *ptr = fopen("game_of_life_gpu.txt", "w+");
  assert(ptr  != NULL);

  unsigned char *buff = (unsigned char *)malloc(mLattice->width *   mLattice->height *  sizeof(unsigned char));
  assert(hipMemcpy(buff, mLattice->buff0, mLattice->width *   mLattice->height *  sizeof(unsigned char),  hipMemcpyDeviceToHost) == hipSuccess);

  for (int j = 1; j < mLattice->height - 1; j++){
      for (int i = 1; i < mLattice->width - 1; i++){
          int k = j * mLattice->width  +  i;
          if (buff[k] == 1)
            fputc('#', ptr);
          else
           fputc(' ', ptr);
      }//end-  for (int i = 0; i < mLattice->width; i++){
      fputc('\n', ptr);
  }//end-for (int j = 0; j < mLattice->height; j++){


  fclose(ptr);
  fprintf(stdout, "\t[OK]\n");
  free(buff);

}
