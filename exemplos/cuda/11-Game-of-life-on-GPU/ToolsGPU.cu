#include "hip/hip_runtime.h"
#include <ToolsGPU.cuh>
#include <ctime>
#include <cstdio>
#define CHECK_ERROR(call) do {                                            \
    if( hipSuccess != call) {                                            \
   fprintf(stderr, "Cuda error in file '%s' in line %i .\n",              \
         __FILE__, __LINE__  );                                           \
         exit(0);                                                         \
} } while (0)


__global__
void kernel(int *buff1, int *buff0){
    unsigned int x      = blockDim.x * blockIdx.x + threadIdx.x,
                 y      = blockDim.y * blockIdx.y + threadIdx.y;
    int cell = 0;
    int myCell = buff0[y * blockDim.x * gridDim.x + x] ;
    int newC = 0;
    //buff1[y * blockDim.x * gridDim.x + x] = 1 - buff0[y * blockDim.x * gridDim.x + x];
    
    if ((x > 0) && ( x < ((blockDim.x * gridDim.x) - 1)) && (y > 0) && (y < ((blockDim.y * gridDim.y) - 1))){
        cell = buff0[(y+1) * blockDim.x * gridDim.x + (x-1)] +  \
               buff0[(y+1) * blockDim.x * gridDim.x + (x)]   +  \
               buff0[(y+1) * blockDim.x * gridDim.x + (x+1)] +  \
               buff0[(y-1) * blockDim.x * gridDim.x + (x-1)] +  \
               buff0[(y-1) * blockDim.x * gridDim.x + (x)]   +  \
               buff0[(y-1) * blockDim.x * gridDim.x + (x+1)] +  \
               buff0[y * blockDim.x * gridDim.x + (x-1)] +  \
               buff0[y * blockDim.x * gridDim.x + (x+1)] ;
    
    }


    if ((cell == 3) && (myCell == 0))   
        newC = 1;

    if ((cell >= 2) && (cell <= 3) && (myCell == 1))
        newC = 1;
    
    
    buff1[y  * blockDim.x * gridDim.x  +  x] = newC;
}

void GameOfLifeGPU(tpLattice *mLattice){
    dim3 threads = dim3(32, 32, 1);
    dim3 blocks  = dim3(mLattice->width / threads.x,  mLattice->height / threads.y, 1);
    kernel<<<blocks, threads>>> (mLattice->buff1, mLattice->buff0);
    CHECK_ERROR(hipDeviceSynchronize());
    
}


void InitRandnessGPU(tpLattice *mLattice, int *buffer, float p){
  memset(buffer, 0x00,  mLattice->width *   mLattice->height *  sizeof(int));
   
  for (int j = 1; j < mLattice->height - 1; j++){
      for (int i = 1; i < mLattice->width - 1; i++){
          int k = j * mLattice->width  +  i;
          float r = (rand() / (float)RAND_MAX);
          if (r <= p)
            buffer[k] = 1;

      }//end-  for (int i = 0; i < mLattice->width; i++){
  }//end-for (int j = 0; j < mLattice->height; j++){
  
  hipMemcpy((void*)mLattice->buff0,  (const void*) buffer,  mLattice->width *   mLattice->height *  sizeof(int), hipMemcpyHostToDevice);
}//end-void InitRandness(tpLattice *mLattice, float p){