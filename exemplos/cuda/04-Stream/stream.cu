    //nvcc -arch=sm_11 -m64 -O3 main.cu -o stream.bin


#include<iostream>
#include<cstdlib>
#include <hip/hip_runtime.h>
#include <cassert>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <ctime>


#define DOMINO 4096
#define BLOCOS        8
#define STREAM        4

#define CHECK_ERROR(call) do {                                                    \
   if( hipSuccess != call) {                                                             \
      std::cerr << std::endl << "CUDA ERRO: " <<                             \
         hipGetErrorString(call) <<  " in file: " << __FILE__                \
         << " in line: " << __LINE__ << std::endl;                               \
         exit(0);                                                                                 \
   } } while (0)


/*
 *************************************************************************
   unsigned int width = gridDim.x * blockDim.x;
   unsigned int height = gridDim.y * blockDim.y;
   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
   unsigned int kn = y * width + x;
 *************************************************************************
 N blocks x M threads  <---- IPC
*/
__global__ void setup_kernel(const unsigned long long seed,  hiprandState *state){
   int x      = blockDim.x * blockIdx.x + threadIdx.x;

   hiprand_init(seed, x, 0, &state[x]);
}

__global__ void uniform_kernel(hiprandState *state, float *h_y){
   
   
     int x      = blockDim.x * blockIdx.x + threadIdx.x;


    h_y[x] =  hiprand_uniform(&state[x]);
    
    

}




using namespace std;
int main (int argc, char **argv){


   hipEvent_t e_Start,
                      e_Stop;

   hipEvent_t e_StartS,
                      e_StopS;
 
   hiprandState       *mStates = NULL;
   
   hipStream_t     *mStreams = NULL;

   float  *h_Mem    = NULL,
           *d_Mem  = NULL;

   
   float elapsedTime = 0.0f,
          elapsedTimeStream = 0.0f,
          acc = 0.0f;
   
   int dominio = DOMINO,
        subdominio = DOMINO / STREAM;
        
   unsigned int qtdeDados = DOMINO * sizeof(float);

                      
   size_t free = 0,
            total = 0;
   
   hipDeviceProp_t deviceProp;                   //Levantar a capacidade do device
   hipGetDeviceProperties(&deviceProp, 0);
   
   
   cout << "\nStream de numeros aleatorios\n";
   
   CHECK_ERROR(hipMemGetInfo(&free, &total));
   cout << "Memoria livre: " << (free / 1024 / 1024)   << " MB\n";
   cout << "Memoria total: " << (total / 1024 / 1024)  << " MB\n";
   cout << "Dominio: " << dominio << endl;
   

     //Reset no device
   CHECK_ERROR(hipDeviceReset());

         //Criando eventos
   CHECK_ERROR(hipEventCreate(&e_Start));
   CHECK_ERROR(hipEventCreate(&e_Stop));

   CHECK_ERROR(hipEventCreate(&e_StartS));
   CHECK_ERROR(hipEventCreate(&e_StopS));

  mStreams = new hipStream_t[STREAM];
   
   for (int i = 0; i < STREAM; i++)
      CHECK_ERROR(hipStreamCreate(&mStreams[i]));

   //Alocando memoria CPU "no-swap"
   CHECK_ERROR(hipHostMalloc(reinterpret_cast<void**> (&h_Mem), qtdeDados, hipHostMallocDefault, hipHostMallocDefault));
   
   //alocando memória em GPU
   CHECK_ERROR(hipMalloc(reinterpret_cast<void**> (&d_Mem), qtdeDados));
   CHECK_ERROR(hipMalloc(reinterpret_cast<void**> (&mStates), dominio * sizeof(hiprandState)));
  
   int blocos = BLOCOS,
        threads = subdominio / BLOCOS;
         
   cout << "Blocos: " << blocos << endl;
   cout << "Threads: " << threads << endl;
   
   assert( threads <= deviceProp.maxThreadsDim[0]);
   
   CHECK_ERROR(hipEventRecord(e_Start, hipEventDefault));
   
    for (int i = 0; i < STREAM; i++){
      CHECK_ERROR(hipEventRecord(e_StartS, mStreams[i]));
      int offset = subdominio * i;
      
      
      setup_kernel<<<blocos, threads,0, mStreams[i] >>>(time (NULL) + offset, mStates+offset);
      uniform_kernel<<<blocos, threads,0, mStreams[i]>>>(mStates+offset, d_Mem + offset);
      CHECK_ERROR(hipMemcpyAsync(h_Mem + offset, d_Mem + offset, subdominio * sizeof(float), hipMemcpyDeviceToHost, mStreams[i] ));
      
   }

   for (int i = 0; i < STREAM; i++){
      CHECK_ERROR( hipStreamSynchronize(mStreams[i]) );
      CHECK_ERROR(hipEventRecord(e_StopS, mStreams[i]));
      CHECK_ERROR(hipEventSynchronize(e_StopS));
      CHECK_ERROR(hipEventElapsedTime(&elapsedTimeStream, e_StartS, e_StopS));
      cout << "Stream: " << i << " tempo: " << (elapsedTimeStream / 1000.0f) << " (s) \n";
      acc += elapsedTimeStream;

   }
   cout << "Acumulado: " << acc / 1000.0f << " (s) \n";

   CHECK_ERROR(hipEventRecord(e_Stop, hipEventDefault));
   CHECK_ERROR(hipEventSynchronize(e_Stop));
   CHECK_ERROR(hipEventElapsedTime(&elapsedTime, e_Start, e_Stop));
   
   cout << "Tempo: " << elapsedTime / 1000.0f << " (s) \n";
   cout <<  endl << "Resultado: "<<  endl;
   for (int i = 0; i < dominio; i++)
         cerr << h_Mem[i]<< endl;
   cerr << endl;
  
   CHECK_ERROR( hipFree(mStates) ); 
   CHECK_ERROR( hipHostFree(h_Mem) );  //Liberando memorias GPU e CPU
   CHECK_ERROR( hipFree(d_Mem) );  //Liberando memorias GPU e CPU
  //  for (int i = 0; i < STREAM; i++)
    //  CHECK_ERROR( cudaStreamDestroy(mStreams[i]) );
   
   delete mStreams;
   CHECK_ERROR( hipEventDestroy (e_Start)  );
   CHECK_ERROR( hipEventDestroy (e_Stop)  );
   CHECK_ERROR( hipEventDestroy (e_StartS)  );
   CHECK_ERROR( hipEventDestroy (e_StopS)  );

   cout << "\nFIM\n";
   return EXIT_SUCCESS;
}
