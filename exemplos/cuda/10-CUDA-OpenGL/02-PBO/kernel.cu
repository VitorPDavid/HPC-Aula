#include "hip/hip_runtime.h"
/*
 *************************************************************************
   unsigned int width = gridDim.x * blockDim.x;
   unsigned int height = gridDim.y * blockDim.y;
   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
   unsigned int kn = y * width + x;
 *************************************************************************
*/

#include <stdio.h>
#include <GPU.h>
#include <App.h>

#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>




__global__
void kernel (uchar4 *out_pixel,
             uchar4 *in_pixel,
             const float ratio)              //Tempo 0)
{
    
    int tx     = threadIdx.x,
        ty     = threadIdx.y,
        width  = gridDim.x * blockDim.x,
        height = gridDim.y * blockDim.y,
        x      = blockDim.x * blockIdx.x + threadIdx.x,
        y      = blockDim.y * blockIdx.y + threadIdx.y,
        kn     = y * width + x;


    float4 pixelA,
           pixelB;


    pixelA.x = (float) in_pixel[kn].x;
    pixelA.y = (float) in_pixel[kn].y;
    pixelA.z = (float) in_pixel[kn].z;
    pixelA.w = (float) in_pixel[kn].w;

    //RGB --> YIQ

    pixelB.x = pixelA.x * 0.299f    + pixelA.y * 0.587f     + pixelA.z * 0.114f;
    pixelB.y = pixelA.x * 0.595716f + pixelA.y * -0.274453f + pixelA.z * -0.321263f;
    pixelB.z = pixelA.x * 0.211456f + pixelA.y * -0.522591f + pixelA.z *  0.311135f;
    //pixelB.w = pixelB.w;



    out_pixel[kn].x = (float) (pixelB.x * ratio) + (pixelA.x * (1.0f - ratio));
    out_pixel[kn].y = (float) (pixelB.y * ratio) + (pixelA.y * (1.0f - ratio));
    out_pixel[kn].z = (float) (pixelB.z * ratio) + (pixelA.z * (1.0f - ratio));
    out_pixel[kn].w = in_pixel[kn].w; //(float) )pixelB.w * ratio) + (pixelA.w * (1.0f - ratio));;
    //out_pixel.x[kn] = in_pixel.x[kn];
    

}
//----------------------------------------------------------------------------------------------------
extern "C" void callCUDA(   uchar4 *out_pixel,
                            uchar4 *in_pixel,
                            int width,
                            int height,
                            const float ratio)
{
    // execute the kernel
    dim3         dGrid,
                 dThreads;

    dGrid.x = width / BSIZE_X;       //BLOCK_SIZE;
    dGrid.y = height / BSIZE_Y;
    dGrid.z = 1;

    dThreads.x =  BSIZE_X;
    dThreads.y =  BSIZE_Y;
    dThreads.z = 1;

  
    CHECK_ERROR(hipDeviceSynchronize());
    kernel<<<dGrid, dThreads>>>(out_pixel, in_pixel, ratio);
    CHECK_ERROR(hipDeviceSynchronize());

}



