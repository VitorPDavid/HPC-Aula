#include "hip/hip_runtime.h"
/*
 *************************************************************************
   unsigned int width = gridDim.x * blockDim.x;
   unsigned int height = gridDim.y * blockDim.y;
   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
   unsigned int kn = y * width + x;
 *************************************************************************
*/

#include <stdio.h>
#include <GPU.h>
#include <App.h>

#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>



__global__ 
void kernelRun (float4 *vertex,
             float *gpu_U1,             //Tempo 1
             float *gpu_U0,
             float *gpu_V)              //Tempo 0)
{
    __shared__ float sharedMem[BSIZE_Y+(2*NPOP)][BSIZE_X+(2*NPOP)];

    int tx     = threadIdx.x,
        ty     = threadIdx.y,
        width  = gridDim.x * blockDim.x,
        height = gridDim.y * blockDim.y,
        x      = blockDim.x * blockIdx.x + threadIdx.x,
        y      = blockDim.y * blockIdx.y + threadIdx.y,
        kn     = y * width + x,
        sX     = tx + NPOP,
        sY     = ty + NPOP;

   float v = VELOCITY,
         old =   gpu_U1[kn];
         
   v = v * v;



   __syncthreads();

   sharedMem[sY][sX] =  gpu_U0[kn] ;

   if (threadIdx.x < NPOP)
   {
      if (blockIdx.x > 0)
         sharedMem[sY][sX-NPOP] = gpu_U0[kn - NPOP] ;
      else
         sharedMem[sY][sX-NPOP] = 0.0f;

      if (blockIdx.x  < (gridDim.x - 1))
         sharedMem[sY][sX+blockDim.x] = gpu_U0[kn + blockDim.x] ;
      else
         sharedMem[sY][sX+blockDim.x] = 0.0f;

   }

   if (threadIdx.y < NPOP)
   {
      if (blockIdx.y > 0)
         sharedMem[sY-NPOP][sX] = gpu_U0[kn - (NPOP*width)];
      else
         sharedMem[sY-NPOP][sX] = 0.0f;

      if (blockIdx.y  < (gridDim.y - 1))
         sharedMem[sY+blockDim.y][sX] = gpu_U0[kn +(blockDim.y*width)] ;
      else
         sharedMem[sY+blockDim.y][sX] = 0.0f;

   }
   __syncthreads();
   
   v *=  gpu_V[y * XPOINTS + x];


  float pX = (sharedMem[sY][sX-2]-(16.0f*(sharedMem[sY][sX-1]+sharedMem[sY][sX+1]))+(30.0f*sharedMem[sY][sX])+sharedMem[sY][sX+2]) * ALPHA * v;
  float pY = (sharedMem[sY-2][sX]-(16.0f*(sharedMem[sY-1][sX]+sharedMem[sY+1][sX]))+(30.0f*sharedMem[sY][sX])+sharedMem[sY+2][sX]) * ALPHA * v;

  gpu_U1[kn] = ((CONSTANT * (pX + pY)) + (2.0f * sharedMem[sY][sX]) - old);


    vertex[kn].x =  (x*4.0f);
    vertex[kn].z =  (y*4.0f);

    vertex[kn].y = gpu_U1[kn]; //+ gpu_U0[kn];
    vertex[kn].w = 1.0f;
 
}


__global__
void kernelPause(float4 *vertex,
             float *gpu_U1,             //Tempo 1
             float *gpu_U0)              //Tempo 0)
{
    int tx     = threadIdx.x,
        ty     = threadIdx.y,
        width  = gridDim.x * blockDim.x,
        height = gridDim.y * blockDim.y,
        x      = blockDim.x * blockIdx.x + threadIdx.x,
        y      = blockDim.y * blockIdx.y + threadIdx.y,
        kn     = y * width + x,
        sX     = tx + NPOP,
        sY     = ty + NPOP;

    vertex[kn].x =  (x*4.0f);
    vertex[kn].z =  (y*4.0f);

    vertex[kn].y = gpu_U1[kn]; // * gpu_R[kn] ; //+ gpu_U0[kn];
    vertex[kn].w = 1.0f;


}
//----------------------------------------------------------------------------------------------------
extern "C" void callCUDARun(float4 *ptrVertice,
                         float *gpu_U0,
                         float *gpu_U1,
                         float *gpu_V,
                         int xVertex,
                         int zVertex)
{
    // execute the kernel
    dim3         dGrid,
                 dThreads;

    dGrid.x = 512 / BSIZE_X;       //BLOCK_SIZE;
    dGrid.y = 512 / BSIZE_Y;
    dGrid.z = 1;

    dThreads.x =  BSIZE_X;
    dThreads.y =  BSIZE_Y;
    dThreads.z = 1;

    
    CHECK_ERROR(hipDeviceSynchronize());
    kernelRun<<<dGrid, dThreads>>>(ptrVertice, gpu_U1, gpu_U0, gpu_V);
    CHECK_ERROR(hipDeviceSynchronize()); 
    
}



extern "C" void callCUDAPause(float4 *ptrVertice,
                         float *gpu_U0,
                         float *gpu_U1,
                         int xVertex,
                         int zVertex)
{
    // execute the kernel
    dim3         dGrid,
                 dThreads;

    dGrid.x = 512 / BSIZE_X;       //BLOCK_SIZE;
    dGrid.y = 512 / BSIZE_Y;
    dGrid.z = 1;

    dThreads.x =  BSIZE_X;
    dThreads.y =  BSIZE_Y;
    dThreads.z = 1;


    CHECK_ERROR(hipDeviceSynchronize());
    kernelPause<<<dGrid, dThreads>>>(ptrVertice, gpu_U1, gpu_U0);
    CHECK_ERROR(hipDeviceSynchronize());

}