#include "hip/hip_runtime.h"
/*
 *************************************************************************
   unsigned int width = gridDim.x * blockDim.x;
   unsigned int height = gridDim.y * blockDim.y;
   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
   unsigned int kn = y * width + x;
 *************************************************************************
*/

#include <stdio.h>
#include <GPU.h>
#include <App.h>

#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>




__global__
void kernel (unsigned char *out_pixel,
             uchar4 *in_pixel)              //Tempo 0)
{
    
    int tx     = threadIdx.x,
        ty     = threadIdx.y,
        width  = gridDim.x * blockDim.x,
        height = gridDim.y * blockDim.y,
        x      = blockDim.x * blockIdx.x + threadIdx.x,
        y      = blockDim.y * blockIdx.y + threadIdx.y,
        kn     = y * width + x;


    float4 pixelA,
           pixelB;


    pixelA.x = (float) in_pixel[kn].x;
    pixelA.y = (float) in_pixel[kn].y;
    pixelA.z = (float) in_pixel[kn].z;
    pixelA.w = (float) in_pixel[kn].w;

    

    out_pixel[kn] = (unsigned char) (pixelB.x);
    

}
//----------------------------------------------------------------------------------------------------
extern "C" void callCUDA(   unsigned char *out_pixel,
                            uchar4 *in_pixel,
                            int width,
                            int height)
{
    // execute the kernel
    dim3         dGrid,
                 dThreads;

    dGrid.x = width / BSIZE_X;       //BLOCK_SIZE;
    dGrid.y = height / BSIZE_Y;
    dGrid.z = 1;

    dThreads.x =  BSIZE_X;
    dThreads.y =  BSIZE_Y;
    dThreads.z = 1;

  
    CHECK_ERROR(hipDeviceSynchronize());
    kernel<<<dGrid, dThreads>>>(out_pixel, in_pixel);
    CHECK_ERROR(hipDeviceSynchronize());

}



