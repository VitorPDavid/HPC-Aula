#include "hip/hip_runtime.h"
/*
 *************************************************************************
   unsigned int width = gridDim.x * blockDim.x;
   unsigned int height = gridDim.y * blockDim.y;
   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
   unsigned int kn = y * width + x;
 *************************************************************************
*/

#include <stdio.h>
#include <GPU.h>
#include <App.h>

//Essas variáveis só podem ser globais!!!!
texture<float, 2>     mTexRef; 
hipChannelFormatDesc mChannelDesc = hipCreateChannelDesc<float>();
hipArray            *mCUMatrizA = NULL;

__global__ 
void kernel (float *vetorB,
             float *vetorA,
             const int colunas,
             const int linhas)
{
    //unsigned int m  = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int n  = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int m = 0;

   // float4 fx = tex2D(mTexRef, x, y );
    //unsigned int n  = blockDim.y * blockIdx.y + threadIdx.y;
    //unsigned int mn = n * width + m;
    vetorB[n] = 0.0f ; //tex2D(mTexRef, 6, 1 ) ;

    for (m = 0; m < colunas; m++)
        vetorB[n] += tex2D(mTexRef, m, n ) * vetorA[m];

    
}
//----------------------------------------------------------------------------------------------------

extern "C" void multiplicaMatrizVetor(float *vetorB,
                                      float *MatrizA,
                                      float *vetorA,
                                      int colunas,
                                      int linhas)
{
    dim3         dGrid,
                 dThreads;

    unsigned int uMemMatrizA  = sizeof(float) * colunas * linhas,
                 uMemVetorA   = sizeof(float) * colunas,
                 uMemVetorB   = sizeof(float) * linhas;

    float        *fGPUVetorB  = NULL,
                 *fGPUVetorA  = NULL;

    Stopwatch sMemoria,
              sGPU;

    dGrid.x = 1;       //BLOCK_SIZE;
    dGrid.y = BLOCK_SIZE;
    dGrid.z = 1;

    dThreads.x = 1;   //colunas / BLOCK_SIZE;
    dThreads.y = linhas  / BLOCK_SIZE;
    dThreads.z = 1;

    FREQUENCY(sMemoria);
    FREQUENCY(sGPU);

    START_STOPWATCH(sMemoria);
    //Aloca memória na GPU
    CHECK_ERROR(hipMallocArray(&mCUMatrizA, &mChannelDesc, colunas, linhas));

    CHECK_ERROR(hipMalloc((void**) &fGPUVetorA, uMemVetorA));
    CHECK_ERROR(hipMalloc((void**) &fGPUVetorB, uMemVetorB));

    //Copiando dados CPU --> GPU
    CHECK_ERROR(hipMemcpyToArray(mCUMatrizA, 0, 0, MatrizA, uMemMatrizA, hipMemcpyHostToDevice));

    CHECK_ERROR(hipMemcpy( fGPUVetorA,  vetorA,  uMemVetorA, hipMemcpyHostToDevice));

    CHECK_ERROR(hipBindTextureToArray(mTexRef, mCUMatrizA));   //Bind da textura

    START_STOPWATCH(sGPU)
    kernel<<<dGrid, dThreads>>>
                            (fGPUVetorB,  fGPUVetorA, colunas, linhas);
    CHECK_ERROR(hipDeviceSynchronize());
    STOP_STOPWATCH(sGPU);

    CHECK_ERROR(hipUnbindTexture(mTexRef));                    //Unbind da textura

    CHECK_ERROR(hipMemcpy(vetorB, fGPUVetorB, uMemVetorB, hipMemcpyDeviceToHost));



    //Desaloca a memória
    CHECK_ERROR(hipFreeArray(mCUMatrizA));
    CHECK_ERROR(hipFree(fGPUVetorA));
    CHECK_ERROR(hipFree(fGPUVetorB));
    STOP_STOPWATCH(sMemoria);

    sMemoria.mElapsedTime -= sGPU.mElapsedTime;

    fprintf(stdout, "\n");
    fprintf(stdout, "\nTotal de memoria alocada na GPU: %u bytes", uMemMatrizA + uMemVetorA +  uMemVetorB);
    fprintf(stdout, "\n            Tempo gasto no processamento: %.4lf (ms) ", sGPU.mElapsedTime);
    fprintf(stdout, "\nTempo gasto com alocacao / copia de mem.: %.4lf (ms) ", sMemoria.mElapsedTime);
    fprintf(stdout, "\n                    Total de tempo gasto: %.4lf (ms) ", sMemoria.mElapsedTime + sGPU.mElapsedTime);
    fprintf(stdout, "\n");


    

}

