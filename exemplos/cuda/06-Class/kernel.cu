#include "hip/hip_runtime.h"
#if !defined(_KERNEL_CU_)
#define _KERNEL_CU_
#include <iostream>
#include <veiculos.cu>


__global__ 
void meuKernel( Veiculo *vet ){

   int indice = blockDim.x * blockIdx.x + threadIdx.x;
   
   if (threadIdx.x == 0)
      vet[indice] = Carro(indice);
   else
      vet[indice] = Onibus(indice);
}



void ExecKernel(int blocos, int threads, Veiculo *vet){
   meuKernel<<<blocos, threads>>> (vet);

}
#endif
