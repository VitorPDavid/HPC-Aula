
#include <hip/hip_runtime.h>
#if !defined(_VEICULOS_CU_)
#define _VEICULOS_CU_
class Veiculo{
   public:
   //Metodos
   __host__ __device__   Veiculo(){};
   __host__ __device__   Veiculo(int id){
      ID     = id + 11;
      x       = 0;
      y       = 0;
      vel    = 0;
      tam = 0;
      vMax = 0;      
   };


   //Atributos
   int ID, x, y, tam, vel, vMax;
};

class Carro : public Veiculo{
public:
   __host__ __device__   Carro(){};
   __host__ __device__   Carro(int id){

         ID     = id+11;
         x       = 0;
         y       = 0;
         vel    = 0;
         tam   = 7; //metros de comprimento
         vMax = 28; //28 metros/s de velocidade maxima
      };

};

class Onibus : public Veiculo{
public:
   __host__ __device__   Onibus(){};
   __host__ __device__   Onibus(int id){
        ID      = id + 11;
         x       = 0;
         y       = 0;
         vel    = 0;
         tam   = 14; //metros de comprimento
         vMax = 23; //28 metros/s de velocidade maxima
      };

};

#endif
